#include "hip/hip_runtime.h"
#include<stdlib.h>
#include<stdio.h>
#include<hip/hip_runtime.h>
#define N 1024*1024

__global__ void dot( int *a, int *b, int *c, int *dot ){
    int tid = threadIdx.x;
    int tid_temp = tid;
	int i;
    int temp =0;
    
    while(tid_temp<N){

        temp += a[tid_temp]*b[tid_temp]; 
        tid_temp += blockDim.x;

    }

    c[tid] = temp;

	
    // synchronize threads in this block
    __syncthreads();
	
    i = blockDim.x/2;
	
    while (i != 0) {
        if (tid < i){
            c[tid] += c[tid + i];   
	}
	__syncthreads();
        i /= 2;
    }
	
    if(tid==0) *dot = c[tid];
}

int main( void ) {
    int *a, *b;
    int *dev_a, *dev_b, *dev_c, *dev_dot;
    int dotCPU = 0;
    int dotGPU;
	
    // allocate the memory on the CPU
    a = (int*)malloc( N * sizeof(int) );
    b = (int*)malloc( N * sizeof(int) );
	
    // allocate the memory on the GPU
    hipMalloc( (void**)&dev_a, N * sizeof(int) );
    hipMalloc( (void**)&dev_b, N * sizeof(int) );
    hipMalloc( (void**)&dev_c, N * sizeof(int) );
    hipMalloc( (void**)&dev_dot, sizeof(int) );

    // fill the arrays 'a' and 'b' on the CPU
    srand ( time(NULL) );
    for (int i=0; i<N; i++) {
        a[i] = rand()%256;
        b[i] = rand()%256;
    }
    // copy the arrays 'a' and 'b' to the GPU
    hipMemcpy( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy( dev_b, b, N * sizeof(int), hipMemcpyHostToDevice );
	
    dot<<<1, N>>>( dev_a, dev_b, dev_c, dev_dot );
    
    // copy the array 'dev_dot' back from the GPU to the CPU
    hipMemcpy( &dotGPU, dev_dot, sizeof(int), hipMemcpyDeviceToHost );
	
    // verify that the GPU did the work we requested
    bool success = true;
    for (int i=0; i<N; i++) {
	dotCPU += a[i] * b[i];
    }
    if (dotCPU != dotGPU) {
        printf( "Error: dotCPU %d != dotGPU %d\n", dotCPU, dotGPU );
        success = false;
    }
    if (success)    printf( "Test pass!\n" );

    // free the memory allocated on the GPU
    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_c );
    hipFree( dev_dot );

    return 0;
}


