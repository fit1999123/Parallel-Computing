#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define N 1024*1024
#define THREADBLOCKS 16
__global__ void vector_add( int *a, int *b, int *c ){
    int tid = threadIdx.x;
    while(tid<N){
        c[tid] = a[tid] + b[tid];
        tid += THREADBLOCKS ;
        }
}

int main( void ) {
    int *a, *b, *c;
    int *dev_a, *dev_b, *dev_c;
	
    // allocate the memory on the CPU
    a = (int*)malloc( N * sizeof(int) );
    b = (int*)malloc( N * sizeof(int) );
    c = (int*)malloc( N * sizeof(int) );
	
    // allocate the memory on the GPU
    hipMalloc( (void**)&dev_a, N * sizeof(int) );
    hipMalloc( (void**)&dev_b, N * sizeof(int) );
    hipMalloc( (void**)&dev_c, N * sizeof(int) );

    // fill the arrays 'a' and 'b' on the CPU
	srand ( time(NULL) );
    for (int i=0; i<N; i++) {
        a[i] = rand()%256;
        b[i] = rand()%256;
    }
    printf("a = %d",&a);
    printf("b = %d",&b);
    // copy the arrays 'a' and 'b' to the GPU
    hipMemcpy( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy( dev_b, b, N * sizeof(int), hipMemcpyHostToDevice );
    
	
    vector_add<<<1, THREADBLOCKS>>>( dev_a, dev_b, dev_c );
    
	
    
    // copy the array 'c' back from the GPU to the CPU
    hipMemcpy( c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost );
	
    // verify that the GPU did the work we requested
    bool success = true;
    for (int i=0; i<N; i++) {
        if ((a[i] + b[i]) != c[i]) {
            printf( "Error:  %d + %d != %d\n", a[i], b[i], c[i] );
            success = false;
        }
    }
    if (success)    printf( "We did it!\n" );

    // free the memory allocated on the GPU
    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_c );

    return 0;
}