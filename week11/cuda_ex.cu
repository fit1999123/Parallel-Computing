#include <stdio.h>



int main( void ) {
    hipDeviceProp_t  prop;
    FILE* fptr;
    fptr=fopen("GPUspec.txt","w");
    if(fptr==NULL){
       printf("FILE cannot be opened.\n");
       exit (1);
    }

    int count;
    hipGetDeviceCount( &count );

    for (int i=0; i< count; i++) {
        hipGetDeviceProperties( &prop, i );
        printf( "   --- General Information for device %d ---\n", i );
        fprintf(fptr, "   --- General Information for device %d ---\n", i );
        printf( "Name:  %s\n", prop.name );
        fprintf(fptr, "Name:  %s\n", prop.name );
        printf( "Compute capability:  %d.%d\n", prop.major, prop.minor );
        fprintf(fptr, "Compute capability:  %d.%d\n", prop.major, prop.minor );
        printf( "Clock rate:  %d\n", prop.clockRate );
        fprintf( fptr,"Clock rate:  %d\n", prop.clockRate );
        printf( "Device copy overlap:  " );
        fprintf( fptr,"Device copy overlap:  " );
        if (prop.deviceOverlap)
            printf( "Enabled\n" );
        else
            printf( "Disabled\n");

        printf( "Kernel execution timeout :  " );
        if (prop.kernelExecTimeoutEnabled)
            printf( "Enabled\n" );
        else
            printf( "Disabled\n" );

        printf( "   --- Memory Information for device %d ---\n", i );
        fprintf(fptr, "   --- Memory Information for device %d ---\n", i );
        printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
        fprintf(fptr, "Total global mem:  %ld\n", prop.totalGlobalMem );
        printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
        fprintf(fptr, "Total constant Mem:  %ld\n", prop.totalConstMem );
        printf( "Max mem pitch:  %ld\n", prop.memPitch );
        fprintf(fptr, "Max mem pitch:  %ld\n", prop.memPitch );
        printf( "Texture Alignment:  %ld\n", prop.textureAlignment );
        fprintf(fptr, "Texture Alignment:  %ld\n", prop.textureAlignment );
        printf( "   --- MP Information for device %d ---\n", i );
        fprintf(fptr, "   --- MP Information for device %d ---\n", i );
        printf( "Multiprocessor count:  %d\n",prop.multiProcessorCount );
        fprintf(fptr, "Multiprocessor count:  %d\n",prop.multiProcessorCount );
        printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
        fprintf(fptr, "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
        printf( "Registers per mp:  %d\n", prop.regsPerBlock );
        fprintf(fptr, "Registers per mp:  %d\n", prop.regsPerBlock );
        printf( "Threads in warp:  %d\n", prop.warpSize );
        fprintf(fptr, "Threads in warp:  %d\n", prop.warpSize );
        printf( "Max threads per block:  %d\n",prop.maxThreadsPerBlock );
        fprintf(fptr, "Max threads per block:  %d\n",prop.maxThreadsPerBlock );
        printf( "Max thread dimensions:  (%d, %d, %d)\n",prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2] );
        fprintf(fptr, "Max thread dimensions:  (%d, %d, %d)\n",prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2] );
        printf( "Max grid dimensions:  (%d, %d, %d)\n",prop.maxGridSize[0], prop.maxGridSize[1],prop.maxGridSize[2] );
        fprintf(fptr, "Max grid dimensions:  (%d, %d, %d)\n",prop.maxGridSize[0], prop.maxGridSize[1],prop.maxGridSize[2] );
        printf( "\n" );
    }
    fclose(fptr);
}
