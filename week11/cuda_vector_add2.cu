#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 16
__global__ void add( int *a, int *b, int *c ){
    int tid = blockIdx.x;
    c[tid] = a[tid] + b[tid];
}


int main( void ) {
    int *a, *b, *c;
    int *dev_a, *dev_b, *dev_c;
	
    // allocate the memory on the CPU
    a = (int*)malloc( N * sizeof(int) );
    b = (int*)malloc( N * sizeof(int) );
    c = (int*)malloc( N * sizeof(int) );
	
    // allocate the memory on the GPU
    hipMalloc( (void**)&dev_a, N * sizeof(int) );
    hipMalloc( (void**)&dev_b, N * sizeof(int) );
    hipMalloc( (void**)&dev_c, N * sizeof(int) );

    // fill the arrays 'a' and 'b' on the CPU
	srand ( time(NULL) );
    for (int i=0; i<N; i++) {
        a[i] = rand()%256;
        b[i] = rand()%256;
    }
    // copy the arrays 'a' and 'b' to the GPU
    hipMemcpy( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy( dev_b, b, N * sizeof(int), hipMemcpyHostToDevice );
    
	
    add<<<N, 1>>>( dev_a, dev_b, dev_c );
    
	
    
    // copy the array 'c' back from the GPU to the CPU
    hipMemcpy( c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost );
    // verify that the GPU did the work we requested
    bool success = true;
    for (int i=0; i<N; i++) {
        if ((a[i] + b[i]) != c[i]) {
            printf( "Error:  %d + %d != %d\n", a[i], b[i], c[i] );
            success = false;
        }
    }
    if (success)    printf( "We did it!\n" );

    // free the memory allocated on the GPU
    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_c );

    return 0;
}