#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define N 100000000
#define THREADBLOCKS 1024

__global__ void vectoradd(int *a,int *b,int *c){

    int tid = threadIdx.x;

    while(tid<N){
    
        c[tid] = a[tid] + b[tid];
        tid+=THREADBLOCKS;
    }
}

int main(){
    int *a = (int*)malloc(N*sizeof(int));
    int *b = (int*)malloc(N*sizeof(int));
    int *c = (int*)malloc(N*sizeof(int));
    int *dev_a, *dev_b, *dev_c;
    hipMalloc( (void**)&dev_a, N * sizeof(int) );
    hipMalloc( (void**)&dev_b, N * sizeof(int) );
    hipMalloc( (void**)&dev_c, N * sizeof(int) );
    srand(time(NULL));
    for (int i=0; i<N; i++) {
        a[i] = rand()%256;
        b[i] = rand()%256;
    }
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipMemcpy( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy( dev_b, b, N * sizeof(int), hipMemcpyHostToDevice );
    hipEventRecord(start, 0);
    vectoradd<<<1,THREADBLOCKS>>>(dev_a,dev_b,dev_c);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipMemcpy( c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost );
    bool success = true;
    for (int i=0; i<N; i++) {
        if ((a[i] + b[i]) != c[i]) {
            printf( "Error:  %d + %d != %d\n", a[i], b[i], c[i] );
            success = false;
        }
    }
    if (success)  {
        printf( "We did it!\n" );
        printf ("Using one block and many threads spend times: %f s\n",time/1000);
    }
    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_c );
    return 0;
}